
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 512


void host_add(int *a, int *b, int *c){
    for (int i = 0 ; i<N;i++)
    c[i] = a[i]+b[i];
}

__global__ void device_add(int *a, int *b, int *c){
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void fill_array(int *data){
    for (int j=0;j<N;j++)
    data[j] = j;
 }

 void print_output(int *a, int *b, int *c){
    for(int idx=0;idx<N;idx++)
		printf("\n %d + %d  = %d",  a[idx] , b[idx], c[idx]);
 }

 int main(void){

    int *a,*b,*c;
    int *d_a,*d_b,*d_c;

     int size =  N * sizeof(int);
     
     // Alloc space for host copies of a, b, c and setup input values
     a = (int *)malloc(size);
     fill_array(a);
     b = (int *)malloc(size);
     fill_array(b);
     c = (int *)malloc(size);

     // Alloc space for device copies of a, b, c
     hipMalloc((void **)&d_a, size);
     hipMalloc((void **)&d_b, size);
     hipMalloc((void **)&d_c, size);

     // Copy inputs to device
     hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
     hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

     //int threads_per_block = 4;
     //int no_of_blocks = N/threads_per_block;	
     device_add<<<N,1>>>(d_a,d_b,d_c);

        // Copy result back to host
        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	print_output(a,b,c);

	free(a); free(b); free(c);
        hipFree(d_a); hipFree(d_b); hipFree(d_c);



	return 0;
 }